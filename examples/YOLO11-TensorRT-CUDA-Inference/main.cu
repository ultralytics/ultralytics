#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include "TRTHandle.h"
#include <string>


float generate_scale(cv::Mat& image, const std::vector<int>& target_size) {
    int origin_w = image.cols;
    int origin_h = image.rows;

    int target_h = target_size[0];
    int target_w = target_size[1];

    float ratio_h = static_cast<float>(target_h) / static_cast<float>(origin_h);
    float ratio_w = static_cast<float>(target_w) / static_cast<float>(origin_w);
    float resize_scale = std::min(ratio_h, ratio_w);
    return resize_scale;
}


float letterbox(cv::Mat &input_image, cv::Mat &output_image, const std::vector<int> &target_size) {
    if (input_image.cols == target_size[1] && input_image.rows == target_size[0]) {
        if (input_image.data == output_image.data) {
            return 1.;
        } else {
            output_image = input_image.clone();
            return 1.;
        }
    }

    float resize_scale = generate_scale(input_image, target_size);
    int new_shape_w = std::round(input_image.cols * resize_scale);
    int new_shape_h = std::round(input_image.rows * resize_scale);
    float padw = (target_size[1] - new_shape_w) / 2.;
    float padh = (target_size[0] - new_shape_h) / 2.;

    int top = std::round(padh - 0.1);
    int bottom = std::round(padh + 0.1);
    int left = std::round(padw - 0.1);
    int right = std::round(padw + 0.1);

    cv::resize(input_image, output_image,
               cv::Size(new_shape_w, new_shape_h),
               0, 0, cv::INTER_AREA);

    cv::copyMakeBorder(output_image, output_image, top, bottom, left, right,
                       cv::BORDER_CONSTANT, cv::Scalar(114., 114., 114));
    return resize_scale;
}

// Undo letterbox for xyxy boxes produced at net_w x net_h (e.g., 640x640)
void undo_letterbox_xyxy(float& x1, float& y1, float& x2, float& y2, int img_w, int img_h, int net_w, int net_h) {
    const float r = std::min(net_w / static_cast<float>(img_w),
        net_h / static_cast<float>(img_h));
    const float new_w = img_w * r, new_h = img_h * r;
    const float dw = (net_w - new_w) * 0.5f;
    const float dh = (net_h - new_h) * 0.5f;

    x1 = (x1 - dw) / r;  y1 = (y1 - dh) / r;
    x2 = (x2 - dw) / r;  y2 = (y2 - dh) / r;

    x1 = std::max(0.f, std::min(x1, static_cast<float>(img_w)));
    y1 = std::max(0.f, std::min(y1, static_cast<float>(img_h)));
    x2 = std::max(0.f, std::min(x2, static_cast<float>(img_w)));
    y2 = std::max(0.f, std::min(y2, static_cast<float>(img_h)));
}


int main(int argc, char **argv) {
    if (argc < 5) {
        std::cerr << "Usage: " << argv[0] << " -input <path_to_input_image> -model <path_to_yolo11_engine>" << std::endl;
        return 1;
    }
    std::string image_path;
    std::string model_path;
    int width = 640;
    int height = 640;
    for (int i = 1; i < argc; ++i) {
        if (std::string(argv[i]) == "-input") {
            image_path = argv[++i];
        } else if (std::string(argv[i]) == "-model") {
            model_path = argv[++i];
        }
    }

    cv::Mat image = cv::imread(image_path);
    cv::Mat input_image;
    
    letterbox(image, input_image, {640, 640});
    if (input_image.empty()) {
        std::cerr << "Error: Something went wrong during letterboxing!" << std::endl;
        return 1;
    }

    // Fetch the model from the specified path
    std::ifstream file(model_path, std::ios::binary | std::ios::ate);
    if (!file)
    {
        auto msg = "Error, unable to open engine file: " + std::string(model_path);
        throw std::runtime_error(msg);
    }
    
    std::streamsize size = file.tellg();
    std::vector<char> modelData(size);
    file.seekg(0, file.beg);
    file.read(reinterpret_cast<char*>(modelData.data()), size);
    file.close();

    if (modelData.empty()) {
        std::cerr << "Failed to read model from file: " << model_path << std::endl;
        std::cerr << "The Engine file path must be given. If it is not present, it can be generated using the build program present in current directory." << std::endl;
        return 1;
    }

    // Sets the device to GPU 0 by default
    checkCudaErrorCode(hipSetDevice(0));

    // Deserialize the engine
    IRuntime* runtime = createInferRuntime(logger);

    std::cout << "Deserializing model from " << model_path << ", size " << size << " bytes" << std::endl;
    ICudaEngine* engine = runtime->deserializeCudaEngine(modelData.data(), modelData.size());

    IExecutionContext *context = engine->createExecutionContext();
    if (!context) {
        std::cerr << "Failed to create execution context from engine" << std::endl;
        return 1;
    }

    // CUDA stream for inference
    hipStream_t stream;
    checkCudaErrorCode(hipStreamCreate(&stream));

    // Initialize I/O buffer vectors
    Buffer buffers(engine, stream);
    std::cout << "Buffer initialized" << std::endl;

    std::vector<void*> buffersVec;
    buffers.getBuffers(buffersVec);

    std::vector<std::string> IOTensorNames;
    buffers.getIOTensorNames(IOTensorNames);

    auto outputLength = buffers.getOutputLength();
    auto inputLength = buffers.getInputLength();

    std::cout << "Input length: " << inputLength << ", Output length: " << outputLength << std::endl;

    for (int i = 0; i < engine->getNbIOTensors(); i++) {
        const auto name = engine->getIOTensorName(i);
        const auto mode = engine->getTensorIOMode(name);
        const auto shape = engine->getTensorShape(name);
        const auto dtype = engine->getTensorDataType(name);
        std::cout << "Tensor " << i << ": " << name
                << ", Mode: " << static_cast<int32_t>(mode)
                << ", Shape rank: " << shape.nbDims;
        for (int j = 0; j < shape.nbDims; j++) {
            std::cout << ", Dim[" << j << "]: " << shape.d[j];
        }
        std::cout << ", Data type: " << static_cast<int32_t>(dtype)
                << std::endl;
    }

    if (! context->setTensorAddress(IOTensorNames[0].c_str(), buffersVec[0])) {
        std::cerr << "Failed to set input tensor address: " << IOTensorNames[0] << std::endl;
        return 1;
    }

    if (! context->setTensorAddress(IOTensorNames[1].c_str(), buffersVec[1])) {
        std::cerr << "Failed to set output tensor address: " << IOTensorNames[1] << std::endl;
        return 1;
    }

    // Preprocess the input image
    // TensorRT model expects input in NCHW format
    cv::Mat input_tensor = cv::dnn::blobFromImage(input_image, 1.0 / 255.0, cv::Size(width, height), cv::Scalar(0, 0, 0), true, false);

    // Copy the input tensor data to the GPU
    checkCudaErrorCode(hipMemcpyAsync(buffersVec[0], input_tensor.data, input_tensor.total() * input_tensor.elemSize(), hipMemcpyHostToDevice, stream));
    std::cout << "Input tensor copied to GPU" << std::endl;

    // Perform inference
    if (! context->enqueueV3(stream)) {
        std::cerr << "Inference failed" << std::endl;
    }

    checkCudaErrorCode(hipStreamSynchronize(stream));

    // Fetch the output from the GPU
    std::vector<float> output_data(outputLength);
    checkCudaErrorCode(hipMemcpyAsync(output_data.data(), buffersVec[1], outputLength * sizeof(float), hipMemcpyDeviceToHost, stream));
    std::cout << "Output tensor copied to CPU" << std::endl;
    checkCudaErrorCode(hipStreamSynchronize(stream));

    // draw bounding boxes on the input image
    for (size_t i = 0; i < output_data.size(); i += 6) {
        auto confidence = output_data[i + 4];
        if (confidence > 0.5 && confidence < 1.0) { // Threshold for confidence
            // The coordinates are supposed to be normalized for the original image size
            int x1 = static_cast<int>(output_data[i]);
            int y1 = static_cast<int>(output_data[i + 1]);
            int x2 = static_cast<int>(output_data[i + 2]);
            int y2 = static_cast<int>(output_data[i + 3]);
            undo_letterbox_xyxy((float&)x1, (float&)y1, (float&)x2, (float&)y2,
				image.cols, image.rows, width, height);
            std::cout << "Detection: [" << x1 << ", " << y1 << ", " << x2 << ", " << y2 << "] with confidence " << confidence << std::endl;
            cv::rectangle(image, cv::Point(x1, y1), cv::Point(x2, y2), cv::Scalar(0, 255, 0), 2);
            std::string label = "Confidence: " + std::to_string(confidence);
            cv::putText(image, label, cv::Point(x1, y1 - 10), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 255, 0), 2);
        }
    }

    cv::imshow("Output", image);
    cv::waitKey(0);

    // Cleanup
    checkCudaErrorCode(hipStreamDestroy(stream));

    return 0;
}